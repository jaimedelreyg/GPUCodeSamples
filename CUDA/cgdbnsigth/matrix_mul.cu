#include <stdio.h>
#include "hipblas.h"
#include "matrix_mul.h"
#include <hipblas.h>

// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B

extern "C"
void Mul(float* A, float* B, int hA, int wA, int wB,
	float* C)
{
	int size;
	const float alpha = 1;
	const float beta = 0;

	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);
	hipMemcpy(Cd, C, size, hipMemcpyHostToDevice);

	//cublasHandle
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	// Compute the execution configuration
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		wA,				/* [m] */ 
		hA,				/* [n] */  
		wB,				/* [k] */ 
		&alpha,				/* alfa */ 
		Ad, wA,				/* A[m][k], num columnas (lda) */ 
		Bd, wB,				/* B[k][n], num columnas (ldb) */
		&beta,				/* beta */
		Cd, wB				/* C[m][n], num columnas (ldc) */
	);

	// Read C from the device
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);

	hipblasDestroy(handle);
}
